/*************************************************************************
<<<<<<< HEAD
 * This file was modified for portability to AMDGPU
 * Copyright (c) 2023-2024, Advanced Micro Devices, Inc. All rights reserved.
 * Copyright (c) 2022-2023, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
=======
 * Copyright (c) 2022-2024, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
>>>>>>> upstream/main
 *
 * See LICENSE for license information.
 ************************************************************************/

#include "extensions.h"
#ifdef NVTE_WITH_USERBUFFERS
#include "comm_gemm_overlap.h"
#endif  // NVTE_WITH_USERBUFFERS

#ifndef USE_ROCM
size_t get_cublasLt_version() {
    return cublasLtGetVersion();
}

<<<<<<< HEAD
=======
size_t get_cudnn_version() {
    return hipdnnGetVersion();
}


>>>>>>> upstream/main
bool userbuf_comm_available() {  // TODO(ksivamani) check on python side
#ifdef NVTE_WITH_USERBUFFERS
    return true;
#else
    return false;
#endif
}
#endif

void placeholder() {}  // TODO(ksivamani) clean this up

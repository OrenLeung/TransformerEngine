#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2022, NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 *
 * See LICENSE for license information.
 ************************************************************************/

#include <transformer_engine/activation.h>
#include <transformer_engine/logging.h>
#include <gtest/gtest.h>
#include <hip/hip_runtime.h>
#ifndef __HIP_PLATFORM_HCC__
#include <hip/hip_bf16.h>
#else
#include <hip/hip_bfloat16.h>
#endif
#include <cmath>
#include <memory>
#include <iostream>
#include <iomanip>
#include <random>
#include <cstring>
#include <type_traits>
#include "../test_common.h"

using namespace transformer_engine;

template <typename IT, typename OT, typename CT>
void compute_ref_gelu_cast(const IT *input_h,
                           OT *output_h,
                           const CT *scale_h,
                           CT *amax_h,
                           const size_t N,
                           const size_t H) {
  CT amax  = 0.;
  CT scale = 1;
  if (std::is_same<OT, test::fp8e4m3>::value ||
      std::is_same<OT, test::fp8e5m2>::value) {
    scale = *scale_h;
  }

  for (size_t i = 0; i < N; i++) {
    for (size_t j = 0; j < H; j++) {
      CT elt = CT(input_h[i * H + j]);
      elt = 0.5f * elt * (1.0f + tanhf(0.79788456F * elt *
                                       (1.0f + 0.044715f * elt * elt)));
      output_h[i * H + j] = OT(scale * elt);
      amax = std::abs(elt) > amax ? std::abs(elt) : amax;
    }
  }

  *amax_h = amax;
}

template <typename IType, typename OType>
void performTestGelu(const size_t N, const size_t H) {
  using namespace test;

  using CType = fp32;

  DType itype = TypeInfo<IType>::dtype;
  DType otype = TypeInfo<OType>::dtype;
  DType ctype = TypeInfo<CType>::dtype;

  Tensor input({ N, H }, itype);
  Tensor output({ N, H }, otype);
  Tensor scale({ 1 }, ctype);
  Tensor amax({ 1 }, ctype);
  Tensor scale_inv({ 1 }, ctype);

  fillUniform(input);
  fillUniform(scale);

  std::unique_ptr<OType[]> ref_output = std::make_unique<OType[]>(N*H);

  nvte_gelu(input.data(), output.data(), scale.data(),
            amax.data(), scale_inv.data(), 0);

  float ref_amax;
  compute_ref_gelu_cast(input.cpu_dptr<IType>(), ref_output.get(),
                        scale.cpu_dptr<float>(),
                        &ref_amax, N, H);

  hipDeviceSynchronize();
  auto err = hipGetLastError();
  ASSERT_EQ(err, hipSuccess) << hipGetErrorString(err);

  if (otype == DType::kFloat8E4M3 || otype == DType::kFloat8E5M2) {
    auto [atol_amax, rtol_amax] = getTolerances(DType::kFloat32);
    compareResults("amax", amax, &ref_amax, atol_amax, rtol_amax);
    float ref_scale_inv = 1.f / (*scale.cpu_dptr<float>());
    compareResults("scale_inv", scale_inv, &ref_scale_inv, atol_amax, rtol_amax);
  }
  auto [atol, rtol] = getTolerances(otype);
  compareResults("output_gelu", output, ref_output.get(), atol, rtol);
}

class GELUTestSuite : public ::testing::TestWithParam<std::tuple<transformer_engine::DType,
                                                                 transformer_engine::DType,
                                                                 std::pair<size_t, size_t>>> {};

TEST_P(GELUTestSuite, TestGELU) {
    using namespace transformer_engine;
    using namespace test;

    const DType input_type = std::get<0>(GetParam());
    const DType output_type = std::get<1>(GetParam());
    const auto size = std::get<2>(GetParam());

    TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(input_type, InputType,
      TRANSFORMER_ENGINE_TYPE_SWITCH_ALL(output_type, OutputType,
        performTestGelu<InputType, OutputType>(size.first, size.second);
      );
    );
}

namespace {

std::vector<std::pair<size_t, size_t>> gelu_test_cases = {{2048, 12288},
                                                          {768, 1024},
                                                          {256, 65536},
                                                          {65536, 128},
                                                          {256, 256},
                                                          {257, 259},
                                                          {128, 128+1}};

}  // namespace

INSTANTIATE_TEST_SUITE_P(
    OperatorTest,
    GELUTestSuite,
    ::testing::Combine(
        ::testing::Values(DType::kFloat32, DType::kBFloat16, DType::kFloat16),
        ::testing::ValuesIn(test::all_fp_types),
        ::testing::ValuesIn(gelu_test_cases)),
    [](const testing::TestParamInfo<GELUTestSuite::ParamType>& info) {
      std::string name = test::typeName(std::get<0>(info.param)) + "X" +
                         test::typeName(std::get<1>(info.param)) + "X" +
                         std::to_string(std::get<2>(info.param).first) + "X" +
                         std::to_string(std::get<2>(info.param).second);
      return name;
    });
